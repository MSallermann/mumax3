#include "hip/hip_runtime.h"

#include "float3.h"

// dst += prefactor * dot(a,b)
extern "C" __global__ void
tangentspaceprojection(
            float* __restrict__ kx, float* __restrict__ ky, float* __restrict__ kz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 k = {kx[i], ky[i], kz[i]};
        float3 m = {mx[i], my[i], mz[i]};

        const float km = dot(k,m);
        
        kx[i] = kx[i] - km*mx[i];
        ky[i] = ky[i] - km*my[i];
        kz[i] = kz[i] - km*mz[i];
    }
}

