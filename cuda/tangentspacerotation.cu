#include "hip/hip_runtime.h"

#include <stdint.h>
#include "float3.h"

// rotates a vector v0 from the tangent space of m0 to the tangent space of m
// saves result in v

extern "C" __global__ void
tangentspacerotation(
        float* __restrict__ vx,  float* __restrict__  vy,  float* __restrict__ vz,
        float* __restrict__ v0x,  float* __restrict__  v0y,  float* __restrict__ v0z,
        float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
        float* __restrict__ m0x,  float* __restrict__  m0y,  float* __restrict__ m0z,
        int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m  = { mx[i],  my[i],  mz[i]};
        float3 m0 = {m0x[i], m0y[i], m0z[i]};
        float3 v0 = {v0x[i], v0y[i], v0z[i]};

        const float SIN = dot(m,v0);
        const float COS = dot(m,m0);

        vx[i] = v0x[i]*COS-m0x[i]*SIN;
        vy[i] = v0y[i]*COS-m0y[i]*SIN;
        vz[i] = v0z[i]*COS-m0z[i]*SIN;

    }
}
