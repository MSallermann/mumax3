#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
zero3(
            float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
            int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        mx[i] = 0.f;
        my[i] = 0.f;
        mz[i] = 0.f;

    }
}
