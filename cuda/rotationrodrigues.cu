#include "hip/hip_runtime.h"
#include <stdint.h>
#include "float3.h"



// Descent energy minimizer
extern "C" __global__ void
rotationrodrigues(
            float* __restrict__ mx,  float* __restrict__  my,  float* __restrict__ mz,
            float* __restrict__ Bx,  float* __restrict__  By,  float* __restrict__ Bz,
            float dt, int N) {

    int i =  ( blockIdx.y*gridDim.x + blockIdx.x ) * blockDim.x + threadIdx.x;
    if (i < N) {

        float3 m = {mx[i], my[i], mz[i]};
        float3 B = {Bx[i], By[i], Bz[i]};

        const float theta = dt*len(B);
        
        // Compute the prefactor efficiently to save time because sin and 1/x are costly.
        // I wonder how much difference this makes.
        float pref;
        if(theta<=1e-2){
            //pref = 1.0 - theta*theta*(1.0-theta*theta/20.0)/6.0;
            // don't divide when using float...
            const float theta2 = theta*theta;
            pref = 1.0 - 0.166667*theta2*(1.0-0.05*theta2); 
        }else
            pref = sin(theta)/theta;
        
        // update m and normalize but check if it is within the sample
        if(!is0(m))
            m = normalized(m*cos(theta) + dt*pref*B); 

        mx[i] = m.x;
        my[i] = m.y;
        mz[i] = m.z;

    }
}
